#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include "hip/hip_runtime_api.h"
#include <unistd.h>
#include <signal.h>
#include <fcntl.h>

volatile bool ready = false;
volatile bool should_stop = false;

const char * ready_fifo = "/tmp/ready"; 


/* 
 * Kernel of dense matrix-matrix multiplication kernel.
 * The algorithm is based on CUDA sgemm code from Vasily Volkov
 * at UC Berkeley.
 */


#define CHECK_ERROR(errorMessage) {                                    \
  hipError_t err = hipGetLastError();                                    \
  if( hipSuccess != err) {                                                \
    fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
	errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
    exit(EXIT_FAILURE);                                                  \
  }                                                                        \
}

// CML x RML = CML, baseline version, 510FLOP/s on Fermi
/* Pseudo code
for i < M ; i += 64   // thread block.x
 for j < N; j += 16   // thread block.y
  for tx = 0; tx < 16; tx++ // thread index x; tile of M loop
  for ty = 0; ty < 4 ; ty++ // thread index y; tile of M loop

  for m < 16; m += 1;
     c[m] = 0.0f

  for k < K; k += 4   // seq

   b[ty][tx] = B[k+ty][j+tx]

   for l < 4; l +=1   // seq
    for m < 16; m +=1 // seq
      c[m] += A[i+ty*16+tx][k+l]+b[l][m]

*/

// Parameters of tile sizes
#define TILE_N 16 
#define TILE_TB_HEIGHT 8
#define TILE_M (TILE_N*TILE_TB_HEIGHT)

__global__ void mysgemmNT( const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta )
{
    // Partial results 
    float c[TILE_N];
    for (int i=0; i < TILE_N; i++)
	c[i] = 0.0f;
    int mid = threadIdx.y * blockDim.x + threadIdx.x; //flattened id
    int m = blockIdx.x * TILE_M + mid;
    int n = blockIdx.y * TILE_N + threadIdx.x;
    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];
    for (int i = 0; i < k; i+=TILE_TB_HEIGHT) {
	float a; 
	b_s[threadIdx.y][threadIdx.x]=B[n + (i+threadIdx.y)*ldb];
	__syncthreads();
	for (int j = 0; j < TILE_TB_HEIGHT; j++) {
	    a = A[m + (i+j)*lda];
	    for (int kk = 0; kk < TILE_N; kk++)
		c[kk] += a * b_s[j][kk];

	}
	__syncthreads();
    }
    int t = ldc*blockIdx.y * TILE_N + m;
    for (int i = 0; i < TILE_N; i++) {
	C[t+i*ldc] = C[t+i*ldc] * beta + alpha * c[i];
    }
}

void regtileSgemm( char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc )
{
  if ((transa != 'N') && (transa != 'n')) {
    std::cerr << "unsupported value of 'transa' in regtileSgemm()" << std::endl;
    return;
  }
  
  if ((transb != 'T') && (transb != 't')) {
    std::cerr << "unsupported value of 'transb' in regtileSgemm()" << std::endl;
    return;
  }
  
  // In this code we assume the matrix sizes are multiple of tile size
  if ((m%TILE_M) || (n%TILE_N)) {
    std::cerr << "unsupported size of matrix. m should be multiple of " << TILE_M
      << "; n should be multiple of " << TILE_N << std::endl;
  }

    /* Create CUDA start & stop events to record total elapsed time of kernel execution */
    hipEvent_t start;
    hipError_t error = hipEventCreate(&start);

    if (error != hipSuccess)
    {   
        fprintf(stderr, "Failed to create start event (error code %s)!\n", 
                                                hipGetErrorString(error));   
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {   
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", 
                                                hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    /* End event creation */ 
    
    
    /* Write to pipe to signal the wrapper script that we are done with data setup */
    char pid[10];
    sprintf(pid, "%d", getpid());

    int fd = open(ready_fifo, O_WRONLY);
    int res = write(fd, pid, strlen(pid));
    close(fd);

    if (res > 0) printf("Parboil sgemm write success to the pipe!\n");
    /* End pipe writing */
    
    
    /* Spin until master tells me to start kernels */
    while (!ready);
    /* End spinning */
    
    
    /* Record the start event and start nvprof profiling */
    hipProfilerStart();
    
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", 
                                                hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    /* End CUDA start records */
 

    dim3 grid( m/TILE_M, n/TILE_N ), threads( TILE_N, TILE_TB_HEIGHT );

    while(!should_stop) 
      mysgemmNT<<<grid, threads>>>( A, lda, B, ldb, C, ldc, k, alpha, beta);

    CHECK_ERROR("mySgemm");

    /* Record and wait for the stop event */
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", 
                                                hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    hipDeviceSynchronize();

    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", 
                                                            hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    hipProfilerStop();
    
    /* End stop CUDA event handling */
    
    
    /* Output total elapsed time */
    float msecTotal = 0.0f;
    // !!! Important: must use this print format, the data processing 
    // script requires this information 
    error = hipEventElapsedTime(&msecTotal, start, stop);
    printf("Total elapsed time: %f ms\n", msecTotal);
    /* End elpased time recording */



}

