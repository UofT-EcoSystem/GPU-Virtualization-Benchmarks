#include "hip/hip_runtime.h"
/*
  baseSetupJs.cu

    Copyright (c) 2015-2019 Johns Hopkins University.  All rights reserved.

    This file is part of the Arioc software distribution.  It is subject to the license terms
    in the LICENSE.txt file found in the top-level directory of the Arioc software distribution.
    The contents of this file, in whole or in part, may only be copied, modified, propagated, or
    redistributed in accordance with the license terms contained in LICENSE.txt.
*/
#include "stdafx.h"

#pragma region CUDA device code and data
/// [kernel] baseSetupJs_Kernel
static __global__ void baseSetupJs_Kernel( const UINT32* const __restrict__ pcnJBuffer, // in: cumulative J-list sizes (one per mate)
                                           const UINT32                     iQ,         // in: index of first QID for the current seed iteration
                                           const UINT32                     nJlists,    // in: number of J lists for the current seed iteration
                                           const UINT32                     npos,       // in: number of seed positions per Q sequence
                                           const UINT32                     sps,        // in: 2: seeding from Q+ and Q-; 1: seeding from Q+ only
                                                 UINT64* const              pDqBuffer   // out: pointer to D-list buffer
                                         )
{
    // compute the 0-based index of the CUDA thread
    const UINT32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // each CUDA thread initializes Dq values for the J list associated with one Q sequence and one seed position
    if( tid >= nJlists )
        return;

    /* Get the QID for the current thread's Q sequence.

       - When only Qf is seeded, there are npos lists for each QID; when Qrc is also seeded, there are
       2*npos lists for each QID.
       - When Qrc is seeded, even-numbered Dq values contain J values for Qf seeds and odd-numbered Dq values
       contain J values for Qrc seeds.
       */
    const UINT32 rcBit = (sps == 2) ? (tid & 1) : 0;
    const UINT32 nSeedsPerQ = npos * sps;
    const UINT32 iQID = iQ + (tid / nSeedsPerQ);    // index of the QID for this CUDA thread's J list
    const UINT32 spos = (tid % nSeedsPerQ) / sps;   // 0-based position of the seed within the Q sequence

    // get the cumulative number of J values preceding the current iteration
    UINT32 iJlist0 = iQ * nSeedsPerQ;
    UINT32 cnJ0 = pcnJBuffer[iJlist0];

    // point to the first D value to be initialized in the current thread
    UINT32 iJlist = iJlist0 + tid;
    UINT64* pDq = pDqBuffer + pcnJBuffer[iJlist] - cnJ0;

    // compute the limiting value for the D-list pointer
    const UINT64* pDlimit = pDqBuffer + pcnJBuffer[iJlist+1] - cnJ0;

    /* Initialize the Dq values for the current thread (Q sequence and seed position).  Yes, this is terrible
        CUDA memory management (i.e., not coalesced) but it only runs once on comparatively little data... */
    UINT32 ij = 0;
    while( pDq < pDlimit )
        *(pDq++) = PACK_DQ( rcBit, iQID, spos, ij++ );
}
#pragma endregion

#pragma region private methods
/// [private] method initConstantMemory
void baseSetupJs::initConstantMemory()
{
}

/// [private] method initSharedMemory
UINT32 baseSetupJs::initSharedMemory()
{
    CRVALIDATOR;

    // provide a hint to the CUDA driver as to how to apportion L1 and shared memory
    CRVALIDATE = hipFuncSetCacheConfig(reinterpret_cast<const void*>( baseSetupJs_Kernel), hipFuncCachePreferL1 );

    return 0;
}

/// [private] method launchKernel
void baseSetupJs::launchKernel( dim3& d3g, dim3& d3b, UINT32 cbSharedPerBlock )
{
    // performance metrics
    InterlockedExchangeAdd( &m_ptum->us.PreLaunch, m_hrt.GetElapsed(true) );

#if TODO_CHOP_WHEN_DEBUGGED
    CDPrint( cdpCD0, "%s: m_nJlists=%d m_nSeedPos=%d cnJ.p=0x%016llx cnJ.Count=%lld",
                        __FUNCTION__, m_nJlists, m_nSeedPos, m_pqb->DBj.cnJ.p, m_pqb->DBj.cnJ.Count );
#endif



    // execute the kernel
    baseSetupJs_Kernel<<< d3g, d3b, cbSharedPerBlock >>>( m_pqb->DBj.cnJ.p,         // in: cumulative per-Q J-list sizes
                                                          m_iQ,                     // in: index of first QID for the current iteration
                                                          m_nJlists,                // in: number of J lists for the current iteration
                                                          m_nSeedPos,               // in: number of seed positions per Q sequence
                                                          m_pab->StrandsPerSeed,    // in: 2: seeding from Q+ and Q-; 1: seeding from Q+ only
                                                          m_pdbb->Diter.p           // out: D list for the current iteration
                                                        );
}
#pragma endregion
