#include "hip/hip_runtime.h"
#include <unistd.h>
#include <inttypes.h>
#include <iostream>
#include <vector>

#include "launch_gemm.h"

//
// CUTLASS includes needed for single-precision GEMM kernel
//
// Defines cutlass::gemm::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/gemm.h"

// Defines cutlass::gemm::SgemmTraits, the structural components 
// for single-precision and integer GEMM
#include "cutlass/gemm/igemm_traits.h"


///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassIgemmNN(
  int M,
  int N,
  int K,
  int8_t alpha,
  int8_t const *A,
  int lda,
  int8_t const *B,
  int ldb,
  int8_t beta,
  int *C,
  int ldc,
  hipStream_t& stream) {

  // Note, GemmTraits<> is a generic template defined for various general matrix product
  // computations within CUTLASS. It is intended to be maximally flexible, and consequently
  // it contains numerous template arguments.
  //
  // To keep the interface manageable, several helpers are defined for plausible compositions
  // including the following example for single-precision GEMM. Typical values are used as
  // default template arguments. See `cutlass/gemm/gemm_traits.h` for more details.
  //
  typedef cutlass::gemm::IgemmTraits<
    cutlass::MatrixLayout::kColumnMajor,   // layout of A matrix
    cutlass::MatrixLayout::kColumnMajor  // layout of B matrix
//    cutlass::Shape<128, 128, 128>, 
//    int8_t
  >
    GemmTraits;

  // Define a CUTLASS GEMM type from a GemmTraits<> instantiation.
  typedef cutlass::gemm::Gemm<GemmTraits> Gemm;

  // Construct and initialize CUTLASS GEMM parameters object.
  //
  // One of CUTLASS's design patterns is to define parameters objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Gemm and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  typename Gemm::Params params;

  int result = params.initialize(
    M,     // GEMM M dimension
    N,     // GEMM N dimension
    K,     // GEMM K dimension
    alpha, // scalar alpha
    A,     // matrix A operand
    lda,
    B,     // matrix B operand
    ldb,
    beta,  // scalar beta
    C,     // source matrix C
    ldc,
    C,     // destination matrix C (may be different memory than source C matrix)
    ldc
  );

  if (result) {
    std::cerr << "Failed to initialize CUTLASS Gemm::Params object." << std::endl;
    return hipErrorInvalidValue;
  }

  // Launch the CUTLASS GEMM kernel.
  Gemm::launch(params, stream);

  // Return any errors associated with the launch or hipSuccess if no error.
  return hipGetLastError();
}



///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

// Kernel to initialize a matrix with small integers for igemm.
__global__ void InitIntMatrix_kernel(
  int8_t *matrix,
  int ldm,
  int rows,
  int columns,
  int seed = 0) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    int offset = i + j * ldm;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    int8_t value = int8_t(((offset + seed) * k % m) - m / 2);

    matrix[offset] = value;
  }
}

/// Simple function to initialize a matrix to arbitrary small integers.
hipError_t InitIntMatrix(int8_t *matrix, int ldm, int rows, int columns, int seed = 0) {

  dim3 block(16, 16);
  dim3 grid(
    (rows + block.x - 1) / block.x,
    (columns + block.y - 1) / block.y
  );

  InitIntMatrix_kernel<<< grid, block >>>(matrix, ldm, rows, columns, seed);

  return hipGetLastError();
}



// Allocates device memory for a matrix then fills with arbitrary small integers.
// Input matrix for igemm
hipError_t AllocateInt8Matrix(int8_t **matrix, int ldm, int rows, int columns, int num_matrices, int seed = 0) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(int8_t) * ldm * columns;
  size_t sizeof_matrices = sizeof_matrix * num_matrices;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrices);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrices);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Initialize matrix elements to arbitrary small integers.
  for (int i = 0; i < num_matrices; ++i) {
      int8_t *p_matrix = &((*matrix)[i * ldm * columns]);
      result = InitIntMatrix(p_matrix, ldm, rows, columns, seed + i);

      if (result != hipSuccess) {
        std::cerr << "Failed to initialize matrix: "
          << hipGetErrorString(result) << std::endl;
        return result;
      }
  }
  
  return result;
}

// Allocates device memory for a matrix then fills with arbitrary small integers.
// Output matrix for igemm
hipError_t AllocateIntMatrix(int **matrix, int ldm, int rows, int columns, int num_matrices, int seed = 0) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(int) * ldm * columns;
  size_t sizeof_matrices = sizeof_matrix * num_matrices;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrices);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrices);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  return result;
}



// Naive reference GEMM computation.
__global__ void ReferenceGemm_kernel(
  int M,
  int N,
  int K,
  int8_t alpha,
  int8_t const *A,
  int lda,
  int8_t const *B,
  int ldb,
  int8_t beta,
  int *C,
  int ldc) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < M && j < N) {
    int accumulator = 0;

    for (int k = 0; k < K; ++k) {
      accumulator += A[i + k * lda] * B[k + j * ldb];
    }

    C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
  }
}

// Reference GEMM computation.
hipError_t ReferenceGemm(
  int M,
  int N,
  int K,
  int8_t alpha,
  int8_t const *A,
  int lda,
  int8_t const *B,
  int ldb,
  int8_t beta,
  int *C,
  int ldc) {

  dim3 block(16, 16);
  dim3 grid(
    (M + block.x - 1) / block.x,
    (N + block.y - 1) / block.y
  );

  ReferenceGemm_kernel<<< grid, block >>>(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);

  return hipGetLastError();
}

// Data setup
hipError_t SetupIgemm(int_mm_info& igemm_info) {
  hipError_t result;

  //
  // Allocate matrices in GPU device memory with arbitrary seeds.
  //

  // Determine number of matrices in the ring buffer
  int max_size = std::max(igemm_info.nitems_A, 
                          std::max(igemm_info.nitems_B, igemm_info.nitems_C));

  igemm_info.num_matrices = (1 << 30) / (max_size * sizeof(int));
  printf("Number of matrices in the buffer: %d\n", igemm_info.num_matrices);



  int seed = time(0);
  printf("Seed for A: %d \n", seed);
  result = AllocateInt8Matrix(&(igemm_info.A), igemm_info.lda, igemm_info.M, 
                          igemm_info.K, igemm_info.num_matrices, seed);

  if (result !=  hipSuccess) {
    return result;
  }


  seed = time(0) >> 3;
  printf("Seed for B: %d \n", seed);
  result = AllocateInt8Matrix(&(igemm_info.B), igemm_info.ldb, igemm_info.K, 
                          igemm_info.N, igemm_info.num_matrices, seed);

  if (result !=  hipSuccess) {
    return result;
  }

  result = AllocateIntMatrix(&(igemm_info.C_cutlass), igemm_info.ldc, igemm_info.M, 
                          igemm_info.N, igemm_info.num_matrices, 101);

  if (result != hipSuccess) {
    return result;
  }

  
  result = AllocateIntMatrix(&(igemm_info.C_reference), igemm_info.ldc, igemm_info.M, 
                            igemm_info.N, igemm_info.num_matrices, 101);

  return result;
}

// Validate kernel results
hipError_t ValidateIgemm(int_mm_info& igemm_info, int niter) {
  //
  // Verify.
  //
  // Launch reference GEMM
  hipError_t result;

  int buffer_idx = 0;

  for (int i = 0; i < niter; i++) {
      int8_t* A_adj = &(igemm_info.A[buffer_idx * igemm_info.nitems_A]);
      int8_t* B_adj = &(igemm_info.B[buffer_idx * igemm_info.nitems_B]);
      int* C_reference_adj = &(igemm_info.C_reference[buffer_idx * igemm_info.nitems_C]);

      result = ReferenceGemm(igemm_info.M, igemm_info.N, igemm_info.K, 
                         igemm_info.alpha, A_adj, igemm_info.lda, B_adj, 
                         igemm_info.ldb, igemm_info.beta, C_reference_adj, 
                         igemm_info.ldc);


      buffer_idx++;

      if(buffer_idx >= igemm_info.num_matrices) buffer_idx = 0;
  }




  if (result != hipSuccess) {
    std::cerr << "Reference IGEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    return result;
  }

  // Copy to host and verify equivalence.
  std::vector<int> host_cutlass(igemm_info.nitems_C * igemm_info.num_matrices, 0);
  std::vector<int> host_reference(igemm_info.nitems_C * igemm_info.num_matrices, 0);

  result = hipMemcpy(host_cutlass.data(), igemm_info.C_cutlass, 
           sizeof(int) * igemm_info.nitems_C * igemm_info.num_matrices, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy CUTLASS IGEMM results: "
      << hipGetErrorString(result) << std::endl;

    return result;
  }

  result = hipMemcpy(host_reference.data(), igemm_info.C_reference, 
           sizeof(int) * igemm_info.nitems_C * igemm_info.num_matrices, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy Reference IGEMM results: "
      << hipGetErrorString(result) << std::endl;

    return result;
  }

  //
  // Test for bit equivalence of results.
  //

  if (host_cutlass != host_reference) {
    std::cerr << "IGEMM CUTLASS results incorrect." << std::endl;

    for (int i = 0; i < 128; ++i) {
      std::cout << host_cutlass[i] << "," << host_reference[i] << std::endl;
    }


    return hipErrorUnknown;
  }
  else {
    std::cout << "Matched!" << std::endl;

  }

  return hipSuccess;

}



