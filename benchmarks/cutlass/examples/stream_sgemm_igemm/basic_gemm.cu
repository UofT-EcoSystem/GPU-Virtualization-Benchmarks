#include "hip/hip_runtime.h"
/*************************************************************************************************
 * Copyright (c) 2017-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example demonstrates how to call a CUTLASS GEMM kernel and provides a naive reference
  matrix multiply kernel to verify its correctness.

  The CUTLASS Gemm template is instantiated in the function CutlassSgemmNN. This is kernel computes
  the general matrix product (GEMM) using single-precision floating-point arithmetic and assumes
  all matrices have column-major layout.

  The threadblock tile size is chosen as 128x128x8 which offers good performance for large matrices.
  See the CUTLASS Parallel for All blog post for more exposition on the tunable parameters available
  in CUTLASS.

  https://devblogs.nvidia.com/cutlass-linear-algebra-cuda/

  Aside from defining and launching the SGEMM kernel, this example does not use any other components
  or utilities within CUTLASS. Such utilities are demonstrated elsewhere in other examples and are
  prevalent in the CUTLASS unit tests.
*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>
#include <algorithm> 
#include <time.h>
#include <inttypes.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <signal.h>
#include <string.h>
#include <fcntl.h>

#include "launch_gemm.h"

#include "hip/hip_runtime_api.h"

volatile bool done = false;

void inline print_current_time_with_ms ()
{
    long ms; 
    time_t s;
    struct timespec spec;

    clock_gettime(CLOCK_REALTIME, &spec);

    s = spec.tv_sec;
    ms = round(spec.tv_nsec / 1.0e6);
    if (ms > 999) {
        s++;
        ms = 0;
    }

    printf("Timestamp: %"PRIdMAX": %03ld\n", (intmax_t)s, ms);
}



///////////////////////////////////////////////////////////////////////////////////////////////////

// call a single-precision and integer CUTLASS GEMM kernel.
hipError_t RunGemm(float_mm_info& sgemm_info, int_mm_info& igemm_info,
                    int niter) {

  hipError_t result;

  const int num_streams = 2;
  hipStream_t streams[num_streams];

  for (int i = 0; i < num_streams; ++i) {
    hipStreamCreate(&streams[i]);
  }

  hipProfilerStart();

  int sbuffer_idx = 0;
  int ibuffer_idx = 0;

  for (int i = 0; i < niter; i++) {
      float* A_adj = &(sgemm_info.A[sbuffer_idx * sgemm_info.nitems_A]);
      float* B_adj = &(sgemm_info.B[sbuffer_idx * sgemm_info.nitems_B]);
      float* C_cutlass_adj = &(sgemm_info.C_cutlass[sbuffer_idx * sgemm_info.nitems_C]);


      result = CutlassSgemmNN(sgemm_info.M, sgemm_info.N, sgemm_info.K, 
                              sgemm_info.alpha, A_adj, sgemm_info.lda, B_adj, 
                              sgemm_info.ldb, sgemm_info.beta, C_cutlass_adj, 
                              sgemm_info.ldc, streams[0]);

      int8_t* iA_adj = &(igemm_info.A[ibuffer_idx * igemm_info.nitems_A]);
      int8_t* iB_adj = &(igemm_info.B[ibuffer_idx * igemm_info.nitems_B]);
      int* iC_cutlass_adj = &(igemm_info.C_cutlass[ibuffer_idx * igemm_info.nitems_C]);

      result = CutlassIgemmNN(igemm_info.M, igemm_info.N, igemm_info.K, 
                              igemm_info.alpha, iA_adj, igemm_info.lda, iB_adj, 
                              igemm_info.ldb, igemm_info.beta, iC_cutlass_adj, 
                              igemm_info.ldc, streams[1]);


      sbuffer_idx++;
      ibuffer_idx++;

      if(sbuffer_idx >= sgemm_info.num_matrices) sbuffer_idx = 0;
      if(ibuffer_idx >= igemm_info.num_matrices) ibuffer_idx = 0;
  }

  hipDeviceSynchronize();
  hipProfilerStop();

  if (result != hipSuccess) {
      std::cerr << "CUTLASS GEMM kernel failed: "
        << hipGetErrorString(result) << std::endl;

      return result;
  }

  return hipSuccess;
}

void cleanup(float_mm_info& sgemm_info, int_mm_info& igemm_info) {
    if (sgemm_info.C_reference != NULL) hipFree(sgemm_info.C_reference);
    if (sgemm_info.C_cutlass != NULL) hipFree(sgemm_info.C_cutlass);
    if (sgemm_info.B != NULL) hipFree(sgemm_info.B);
    if (sgemm_info.A != NULL) hipFree(sgemm_info.A);

    if (igemm_info.C_reference != NULL) hipFree(igemm_info.C_reference);
    if (igemm_info.C_cutlass != NULL) hipFree(igemm_info.C_cutlass);
    if (igemm_info.B != NULL) hipFree(igemm_info.B);
    if (igemm_info.A != NULL) hipFree(igemm_info.A);

}

///////////////////////////////////////////////////////////////////////////////////////////////////



/// Entry point to basic_gemm example.
//
// usage:
//
//   00_basic_gemm <M> <N> <K> <alpha> <beta>
//
int main(int argc, const char *arg[]) {
  print_current_time_with_ms ();

  //
  // Parse the command line to obtain GEMM dimensions and scalar values.
  //

  // GEMM problem dimensions.
  int problem[4] = { 128, 128, 128, 10 };

  for (int i = 1; i < argc && i < 5; ++i) {
    std::stringstream ss(arg[i]);
    ss >> problem[i - 1];
  }

  // Scalars used for linear scaling the result of the matrix product.
  float scalars[2] = { 1, 0 };

  for (int i = 5; i < argc && i < 7; ++i) {
    std::stringstream ss(arg[i]);
    ss >> scalars[i - 5];
  }

  printf("Run CUTLASS matrix multiply A(%d,%d) B(%d,%d) for %d times\n", 
          problem[0], problem[2], problem[2], problem[1], problem[3]);

  float_mm_info sgemm_info;

  sgemm_info.M = problem[0];
  sgemm_info.N = problem[1];
  sgemm_info.K = problem[2];
  sgemm_info.niter = problem[3];

  sgemm_info.lda = sgemm_info.M;
  sgemm_info.ldb = sgemm_info.K;
  sgemm_info.ldc = sgemm_info.M;

  sgemm_info.nitems_A = sgemm_info.lda * sgemm_info.K;
  sgemm_info.nitems_B = sgemm_info.ldb * sgemm_info.N;
  sgemm_info.nitems_C = sgemm_info.ldc * sgemm_info.N;

  sgemm_info.alpha = scalars[0];
  sgemm_info.beta = scalars[1];


  // GEMM problem dimensions.

  for (int i = 7; i < argc && i < 11; ++i) {
    std::stringstream ss(arg[i]);
    ss >> problem[i - 7];
  }

  // Scalars used for linear scaling the result of the matrix product.
  int8_t iscalars[2] = { 1, 0 };

  for (int i = 11; i < argc && i < 13; ++i) {
    std::stringstream ss(arg[i]);
    ss >> iscalars[i - 11];
  }

  printf("Run CUTLASS INT matrix multiply A(%d,%d) B(%d,%d) for %d times\n", 
          problem[0], problem[2], problem[2], problem[1], problem[3]);


  int_mm_info igemm_info;

  igemm_info.M = problem[0];
  igemm_info.N = problem[1];
  igemm_info.K = problem[2];
  igemm_info.niter = problem[3];

  igemm_info.lda = igemm_info.M;
  igemm_info.ldb = igemm_info.K;
  igemm_info.ldc = igemm_info.M;

  igemm_info.nitems_A = igemm_info.lda * igemm_info.K;
  igemm_info.nitems_B = igemm_info.ldb * igemm_info.N;
  igemm_info.nitems_C = igemm_info.ldc * igemm_info.N;

  igemm_info.alpha = iscalars[0];
  igemm_info.beta = iscalars[1];

  hipError_t result;
  
  result = SetupSgemm(sgemm_info);

  if (result != hipSuccess) {
    std::cout << "Failed sgemm input setup." << std::endl;

    cleanup(sgemm_info, igemm_info);

    return -1;
  }

  result = SetupIgemm(igemm_info);

  if (result != hipSuccess) {
    std::cout << "Failed igemm input setup." << std::endl;

    cleanup(sgemm_info, igemm_info);

    return -1;
  }


  result = RunGemm(sgemm_info, igemm_info, sgemm_info.niter);

  if (result != hipSuccess) {
    std::cout << "Failed gemm run." << std::endl;

    cleanup(sgemm_info, igemm_info);

    return -1;
  }

  result = ValidateSgemm(sgemm_info, sgemm_info.niter);

  if (result != hipSuccess) {
    std::cout << "Failed sgemm validation." << std::endl;

    cleanup(sgemm_info, igemm_info);

    return -1;
  }


  result = ValidateIgemm(igemm_info, sgemm_info.niter);


  cleanup(sgemm_info, igemm_info);
  
  if (result == hipSuccess) {
    std::cout << "Passed." << std::endl;
  }

  // Exit.
  return result == hipSuccess ? 0 : -1;
}

