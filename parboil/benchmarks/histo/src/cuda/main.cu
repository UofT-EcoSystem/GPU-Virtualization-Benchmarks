#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/


#include <parboil.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <fcntl.h>


#include "hip/hip_runtime_api.h"
#include "util.h"

volatile bool ready = false;
volatile bool should_stop = false;
const char * ready_fifo = "/tmp/ready"; 

void start_handler(int sig) {
    ready = true;
}

void stop_handler(int sig) {
    should_stop = true;
}


__global__ void histo_prescan_kernel (
        unsigned int* input,
        int size,
        unsigned int* minmax);

__global__ void histo_main_kernel (
        uchar4 *sm_mappings,
        unsigned int num_elements,
        unsigned int sm_range_min,
        unsigned int sm_range_max,
        unsigned int histo_height,
        unsigned int histo_width,
        unsigned int *global_subhisto,
        unsigned int *global_histo,
        unsigned int *global_overflow);

__global__ void histo_intermediates_kernel (
        uint2 *input,
        unsigned int height,
        unsigned int width,
        unsigned int input_pitch,
        uchar4 *sm_mappings);

__global__ void histo_final_kernel (
        unsigned int sm_range_min,
        unsigned int sm_range_max,
        unsigned int histo_height,
        unsigned int histo_width,
        unsigned int *global_subhisto,
        unsigned int *global_histo,
        unsigned int *global_overflow,
        unsigned int *final_histo);

/******************************************************************************
* Implementation: GPU
* Details:
* in the GPU implementation of histogram, we begin by computing the span of the
* input values into the histogram. Then the histogramming computation is carried
* out by a (BLOCK_X, BLOCK_Y) sized grid, where every group of Y (same X)
* computes its own partial histogram for a part of the input, and every Y in the
* group exclusively writes to a portion of the span computed in the beginning.
* Finally, a reduction is performed to combine all the partial histograms into
* the final result.
******************************************************************************/

int main(int argc, char* argv[]) {
  if (signal(SIGUSR1, start_handler) < 0)
      perror("Signal error");

  if (signal(SIGUSR2, stop_handler) < 0)
      perror("Signal error");

  struct pb_TimerSet timers;
  struct pb_Parameters *parameters;

  parameters = pb_ReadParameters(&argc, argv);
  if (!parameters)
    return -1;

  if(!parameters->inpFiles[0]){
    fputs("Input file expected\n", stderr);
    return -1;
  }
  
  char *prescans = "PreScanKernel";
  char *postpremems = "PostPreMems";
  char *intermediates = "IntermediatesKernel";
  char *mains = "MainKernel";
  char *finals = "FinalKernel";

  pb_InitializeTimerSet(&timers);
  
  pb_AddSubTimer(&timers, prescans, pb_TimerID_KERNEL);
  pb_AddSubTimer(&timers, postpremems, pb_TimerID_KERNEL);
  pb_AddSubTimer(&timers, intermediates, pb_TimerID_KERNEL);
  pb_AddSubTimer(&timers, mains, pb_TimerID_KERNEL);
  pb_AddSubTimer(&timers, finals, pb_TimerID_KERNEL);
  
  pb_SwitchToTimer(&timers, pb_TimerID_IO);

  int numIterations;
  if (argc >= 2){
    numIterations = atoi(argv[1]);
  } else {
    fputs("Expected at least one command line argument\n", stderr);
    return -1;
  }

  unsigned int img_width, img_height;
  unsigned int histo_width, histo_height;

  FILE* f = fopen(parameters->inpFiles[0],"rb");
  int result = 0;

  result += fread(&img_width,    sizeof(unsigned int), 1, f);
  result += fread(&img_height,   sizeof(unsigned int), 1, f);
  result += fread(&histo_width,  sizeof(unsigned int), 1, f);
  result += fread(&histo_height, sizeof(unsigned int), 1, f);

  if (result != 4){
    fputs("Error reading input and output dimensions from file\n", stderr);
    return -1;
  }

  unsigned int* img = (unsigned int*) malloc (img_width*img_height*sizeof(unsigned int));
  unsigned char* histo = (unsigned char*) calloc (histo_width*histo_height, sizeof(unsigned char));

  result = fread(img, sizeof(unsigned int), img_width*img_height, f);

  fclose(f);

  if (result != img_width*img_height){
    fputs("Error reading input array from file\n", stderr);
    return -1;
  }

  int even_width = ((img_width+1)/2)*2;
  unsigned int* input;
  unsigned int* ranges;
  uchar4* sm_mappings;
  unsigned int* global_subhisto;
  unsigned short* global_histo;
  unsigned int* global_overflow;
  unsigned char* final_histo;

  hipMalloc((void**)&input           , even_width*(((img_height+UNROLL-1)/UNROLL)*UNROLL)*sizeof(unsigned int));
  hipMalloc((void**)&ranges          , 2*sizeof(unsigned int));
  hipMalloc((void**)&sm_mappings     , img_width*img_height*sizeof(uchar4));
  hipMalloc((void**)&global_subhisto , BLOCK_X*img_width*histo_height*sizeof(unsigned int));
  hipMalloc((void**)&global_histo    , img_width*histo_height*sizeof(unsigned short));
  hipMalloc((void**)&global_overflow , img_width*histo_height*sizeof(unsigned int));
  hipMalloc((void**)&final_histo     , img_width*histo_height*sizeof(unsigned char));

  hipMemset(final_histo , 0 , img_width*histo_height*sizeof(unsigned char));

  for (int y=0; y < img_height; y++){
    hipMemcpy(&(((unsigned int*)input)[y*even_width]),&img[y*img_width],img_width*sizeof(unsigned int), hipMemcpyHostToDevice);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);


  /* Create CUDA start & stop events to record total elapsed time of kernel execution */
  hipEvent_t start;
  hipError_t error = hipEventCreate(&start);

  if (error != hipSuccess)
  {   
      fprintf(stderr, "Failed to create start event (error code %s)!\n", 
                                              hipGetErrorString(error));   
      exit(EXIT_FAILURE);
  }

  hipEvent_t stop;
  error = hipEventCreate(&stop);

  if (error != hipSuccess)
  {   
      fprintf(stderr, "Failed to create stop event (error code %s)!\n", 
                                              hipGetErrorString(error));
      exit(EXIT_FAILURE);
  }

  /* End event creation */ 
  
  
  /* Write to pipe to signal the wrapper script that we are done with data setup */
  char pid[10];
  sprintf(pid, "%d", getpid());

  int fd = open(ready_fifo, O_WRONLY);
  int res = write(fd, pid, strlen(pid));
  close(fd);

  if (res > 0) printf("Parboil spmv write success to the pipe!\n");
  /* End pipe writing */
  
  
  /* Spin until master tells me to start kernels */
  while (!ready);
  /* End spinning */
  
  
  /* Record the start event and start nvprof profiling */
  hipProfilerStart();
  
  error = hipEventRecord(start, NULL);

  if (error != hipSuccess)
  {
      fprintf(stderr, "Failed to record start event (error code %s)!\n", 
                                              hipGetErrorString(error));
      exit(EXIT_FAILURE);
  }
  
  /* End CUDA start records */
 


  while (!should_stop){ 
    //for (int iter = 0; iter < numIterations; iter++) {
    unsigned int ranges_h[2] = {UINT32_MAX, 0};

    hipMemcpy(ranges,ranges_h, 2*sizeof(unsigned int), hipMemcpyHostToDevice);
    
    pb_SwitchToSubTimer(&timers, prescans , pb_TimerID_KERNEL);

    histo_prescan_kernel<<<dim3(PRESCAN_BLOCKS_X),dim3(PRESCAN_THREADS)>>>((unsigned int*)input, img_height*img_width, ranges);
    
    pb_SwitchToSubTimer(&timers, postpremems , pb_TimerID_KERNEL);

    hipMemcpy(ranges_h,ranges, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipMemset(global_subhisto,0,img_width*histo_height*sizeof(unsigned int));
    
    pb_SwitchToSubTimer(&timers, intermediates, pb_TimerID_KERNEL);

    histo_intermediates_kernel<<<dim3((img_height + UNROLL-1)/UNROLL), dim3((img_width+1)/2)>>>(
                (uint2*)(input),
                (unsigned int)img_height,
                (unsigned int)img_width,
                (img_width+1)/2,
                (uchar4*)(sm_mappings)
    );
    
    pb_SwitchToSubTimer(&timers, mains, pb_TimerID_KERNEL);
    
    
    histo_main_kernel<<<dim3(BLOCK_X, ranges_h[1]-ranges_h[0]+1), dim3(THREADS)>>>(
                (uchar4*)(sm_mappings),
                img_height*img_width,
                ranges_h[0], ranges_h[1],
                histo_height, histo_width,
                (unsigned int*)(global_subhisto),
                (unsigned int*)(global_histo),
                (unsigned int*)(global_overflow)    
    );
    
    pb_SwitchToSubTimer(&timers, finals, pb_TimerID_KERNEL);
    
    histo_final_kernel<<<dim3(BLOCK_X*3), dim3(512)>>>(
                ranges_h[0], ranges_h[1],
                histo_height, histo_width,
                (unsigned int*)(global_subhisto),
                (unsigned int*)(global_histo),
                (unsigned int*)(global_overflow),
                (unsigned int*)(final_histo)
    );
  }

  /* Record and wait for the stop event */
  error = hipEventRecord(stop, NULL);

  if (error != hipSuccess)
  {
      fprintf(stderr, "Failed to record stop event (error code %s)!\n", 
                                              hipGetErrorString(error));
      exit(EXIT_FAILURE);
  }
  
  hipDeviceSynchronize();

  error = hipEventSynchronize(stop);

  if (error != hipSuccess)
  {
      fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", 
                                                          hipGetErrorString(error));
      exit(EXIT_FAILURE);
  }
  
  hipProfilerStop();
  
  /* End stop CUDA event handling */
    
    
  /* Output total elapsed time */
  float msecTotal = 0.0f;
  // !!! Important: must use this print format, the data processing 
  // script requires this information 
  error = hipEventElapsedTime(&msecTotal, start, stop);
  printf("Total elapsed time: %f ms\n", msecTotal);
  /* End elpased time recording */


  hipMemcpy(histo,final_histo, histo_height*histo_width*sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(input);
  hipFree(ranges);
  hipFree(sm_mappings);
  hipFree(global_subhisto);
  hipFree(global_histo);
  hipFree(global_overflow);
  hipFree(final_histo);

  if (parameters->outFile) {
    dump_histo_img(histo, histo_height, histo_width, parameters->outFile);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  free(img);
  free(histo);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  printf("\n");
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(parameters);
  
  pb_DestroyTimerSet(&timers);

  return 0;
}
