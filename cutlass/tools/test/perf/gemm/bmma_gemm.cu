/***************************************************************************************************
 * Copyright (c) 2017-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/// \file {nv-internal-release}

#if (defined(__HIPCC__) && (!defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 750))
#pragma warning( disable : 4503)
////////////////////////////////////////////////////////////////////////////////////////////////////

#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/bmma_gemm_traits.h"
#include "tools/test/perf/cutlass_perf_test.h"
#include "tools/test/perf/gemm/gemm_profiler.h"
#include "tools/test/perf/gemm/cutlass_dispatch.h"
#include "tools/test/perf/gemm/gemm_perf_testbed.h"

////////////////////////////////////////////////////////////////////////////////////////////////////

template<typename Traits>
struct BmmaGemmDispatch {

  typedef cutlass::gemm::Gemm<Traits> Gemm;

  typedef typename Gemm::Params Params;

  /// Indicate warp-level GEMM
  static bool const kThreadMultiplyAdd = false;

  static bool const kRunCuBLAS = false;

  static cutlass::MatrixLayout::Kind const kLayoutA = Traits::kLayoutA;
  static cutlass::MatrixLayout::Kind const kLayoutB = Traits::kLayoutB;

  //
  // Data members
  //

  /// Params argument
  Params params;

  //
  // Methods
  //

  BmmaGemmDispatch() {}

  /// Initializes params object
  BmmaGemmDispatch(int m, int n, int k, int alpha,
                   cutlass::Vector<cutlass::bin1_t, 32> const* d_a, int lda,
                   cutlass::Vector<cutlass::bin1_t, 32> const* d_b, int ldb, int beta,
                   int const* d_c, int ldc, int* d_d, int ldd) {

    params.initialize(m, n, k * 32, alpha, d_a, lda, d_b, ldb, beta, d_c, ldc, d_d, ldd);
  }

  /// Initializes params object
  BmmaGemmDispatch(Params const& _params) : params(_params) {}

  /// Launches kernel
  hipError_t operator()() { return Gemm::launch(params); }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace perf {

////////////////////////////////////////////////////////////////////////////////////////////////////

int profile_bmma_gemm(TestbenchOutput<GemmProblem> &output, TestbenchOptions const &options, Config const &config) {
  typedef perf::GemmProfiler<cutlass::Vector<cutlass::bin1_t, 32>, cutlass::Vector<cutlass::bin1_t, 32>, int, int, int> GemmProfiler;

  int results = 0;

  {

    typedef cutlass::gemm::BmmaGemmTraits<cutlass::Shape<1024, 128, 128>,
                                          cutlass::Shape<1024, 32, 32>,
                                          cutlass::MatrixLayout::kRowMajor,
                                          cutlass::MatrixLayout::kColumnMajor>
      BmmaGemmTraits;

    typedef BmmaGemmDispatch<BmmaGemmTraits> Dispatch;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "bmma_gemm_tn", options, config);
  }

  return results;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

struct BmmaGemmRegistrar {
  BmmaGemmRegistrar() { RegisterGemmProfileFunc(profile_bmma_gemm); }
};

volatile BmmaGemmRegistrar _BmmaGemmRegistrar;

} // namespace perf

#endif // if (defined(__HIPCC__) && (!defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 750)
