#include "hip/hip_runtime.h"
#include <unistd.h>
#include "rodinia_common.h"

__global__ void hotspotOpt1(float *p, float* tIn, float *tOut, float sdc,
        int nx, int ny, int nz,
        float ce, float cw, 
        float cn, float cs,
        float ct, float cb, 
        float cc) 
{
    float amb_temp = 80.0;

    int i = blockDim.x * blockIdx.x + threadIdx.x;  
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int c = i + j * nx;
    int xy = nx * ny;

    int W = (i == 0)        ? c : c - 1;
    int E = (i == nx-1)     ? c : c + 1;
    int N = (j == 0)        ? c : c - nx;
    int S = (j == ny-1)     ? c : c + nx;

    float temp1, temp2, temp3;
    temp1 = temp2 = tIn[c];
    temp3 = tIn[c+xy];
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;

    for (int k = 1; k < nz-1; ++k) {
        temp1 = temp2;
        temp2 = temp3;
        temp3 = tIn[c+xy];
        tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
            + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    temp1 = temp2;
    temp2 = temp3;
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    return;
}

extern bool set_and_check(int uid, bool start);

void hotspot_opt1(float *p, float *tIn, float *tOut,
        int nx, int ny, int nz,
        float Cap, 
        float Rx, float Ry, float Rz, 
        float dt, int numiter,
        int uid, hipStream_t & stream)
{
    float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;
    ce = cw =stepDivCap/ Rx;
    cn = cs =stepDivCap/ Ry;
    ct = cb =stepDivCap/ Rz;

    cc = 1.0 - (2.0*ce + 2.0*cn + 3.0*ct);

    size_t s = sizeof(float) * nx * ny * nz;  
    float  *tIn_d, *tOut_d, *p_d;
    hipMalloc((void**)&p_d,s);
    hipMalloc((void**)&tIn_d,s);
    hipMalloc((void**)&tOut_d,s);
    hipMemcpyAsync(tIn_d, tIn, s, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(p_d, p, s, hipMemcpyHostToDevice, stream);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(hotspotOpt1), hipFuncCachePreferL1);

    dim3 block_dim(64, 4, 1);
    dim3 grid_dim(nx / 64, ny / 4, 1);

    set_and_check(uid, true);
    while (!set_and_check(uid, true)) {
      usleep(100);
    }

    bool can_exit = false;

    long long start = get_time();

    while (!can_exit) {
//    for (int i = 0; i < numiter; ++i) {
        hotspotOpt1<<<grid_dim, block_dim, 0, stream>>>
            (p_d, tIn_d, tOut_d, stepDivCap, nx, ny, nz, ce, cw, cn, cs, ct, cb, cc);
        float *t = tIn_d;
        tIn_d = tOut_d;
        tOut_d = t;

        hipDeviceSynchronize();
        can_exit = set_and_check(uid, false);
    }


    hipDeviceSynchronize();
    long long stop = get_time();
    float time = (float)((stop - start)/(1000.0 * 1000.0));
    printf("Time: %.3f (s)\n",time);
    hipMemcpyAsync(tOut, tOut_d, s, hipMemcpyDeviceToHost, stream);
    hipDeviceSynchronize();

    hipFree(p_d);
    hipFree(tIn_d);
    hipFree(tOut_d);
    return;
}

