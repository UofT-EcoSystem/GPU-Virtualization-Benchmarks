/*********************************************************
 * Wrapper to create cross-product pairs of benchmarks
 *                Created by Serina Tan 
 *                     Apr 5, 2019 
 * *******************************************************
 * ******************************************************/


// c++ includes
#include <stdio.h>
#include <unistd.h>
#include <iostream>
#include <sstream>
#include <functional>
#include <string>
#include <fstream>
#include <thread>
#include <mutex>
#include <cassert>
#include <vector>
#include <functional>

// cuda include
// #include "hip/hip_runtime.h"

// user includes
#include "parboil/benchmarks/interface.h"
#include "cutlass/interface.h"

std::vector<bool> done_flags;
std::mutex lock_flag;

bool set_and_check(int uid) {
  // this function is guarded by the mutex
  std::lock_guard<std::mutex> guard(lock_flag);

  if (uid < done_flags.size()) {
    done_flags[uid] = true;
  }

  
  for (auto f : done_flags) {
    if (!f) return false;
  }

  return true;
}

void shared_push() {
  // this function is guarded by the mutex
  std::lock_guard<std::mutex> guard(lock_flag);
  done_flags.push_back(false);
}


void invoke(int uid, std::string kernel_arg)
{
  // split string into argv
  std::vector<std::string> string_argv;
  std::stringstream ss(kernel_arg);
  std::string token;
  while (std::getline(ss, token, ' ')) {
    if (token.length() > 0)
      string_argv.push_back(token);
  }

  // assign each char array
  int argc = string_argv.size();
  char* argv[argc];
  // this vector maintains the original char array pointers
  // cuz the main function will modify the argv
  // this is a sketchy solution
  std::vector<char*> to_free;
  assert(argc > 0);
  for (int i = 0; i < string_argv.size(); i++) {
    argv[i] = new char[string_argv[i].length()+1];
    strcpy (argv[i], string_argv[i].c_str());
    to_free.push_back(argv[i]);
  }

  // select the right benchmark symbol
  std::function<int(int,char**,int)> func = NULL;
  if (strcmp(argv[0], "parb_sgemm") == 0) {
    std::cout << "main: parboil sgemm" << std::endl;
#ifdef PARBOIL_SGEMM
    func = main_sgemm;
#endif
  } else if (strcmp(argv[0], "parb_stencil") == 0) {
    std::cout << "main: parboil stencil" << std::endl;
#ifdef PARBOIL_STENCIL
    func = main_stencil;
#endif
  } else if (strcmp(argv[0], "cut_sgemm") == 0) {
    std::cout << "main: cutlass sgemm" << std::endl;
#ifdef CUT_SGEMM
    func = main_sgemm;
#endif
  } else if (strcmp(argv[0], "cut_wmma") == 0) {
    std::cout << "main: cutlass wmma" << std::endl;
#ifdef CUT_WMMA
    func = main_wmma;
#endif
  } 
  else {
    std::cout << "Warning: No matching kernels for " << argv[0] << std::endl;
  }

  if (func == NULL) {
    std::cout << "Empty function pointer. Check your compile defines." << std::endl;
    exit(1);
  }

  // invoke the real function
  func(argc, argv, uid);

  // cleanup the char arrays
  for (auto carray: to_free) {
    delete carray;
  }

}


int main(int argc, char** argv) {
  if (argc < 2 || argv[1] == "-h") {
    std::cout << "Usage: ";
    std::cout << "./driver RUNFILE1 [RUNFILE2]" << std::endl;
    abort();
  } 

  std::vector<std::string> args;
  for (int i = 1; i < argc; ++i) {
    char* filename = argv[i];

    // extract run arguments from file
    // expect a single line file
    std::string line;
    std::ifstream file (filename);
    if (file.is_open() && std::getline (file,line))
    {
      args.push_back(line);
      file.close();
    } else {
      std::cout << "Error reading file: " << filename << std::endl;
    }
  }

  // spawn threads to invoke separate kernels
  std::thread ts[args.size()];
  for (int i = 0; i < args.size(); i++) {
    shared_push();
    ts[i] = std::thread(invoke, i, args[i]);
  }

  // join threads
  for (auto & t : ts) {
    t.join();
  }

  // sanity check: all flags should now be true
  for (auto f : done_flags) {
    if (!f) std::cout << "Some thread did not set flag to true!!!!" << std::endl;
  }

  hipDeviceSynchronize();

  return 0;


//  // run the kernels
//  if (strncmp("1",argv[1],1) == 0) {
//    hipStream_t stream_A;
//    hipStreamCreate(&stream_A);
//
//    // grab kernel launch and exit function calls from benchmark A and B
//    std::function<int(const int, hipStream_t &)> kernel_A;
//    std::function<void(void)> cleanup_A;
//
//    invoke(A_str, argc_A, &(argv[A_idx]), kernel_A, cleanup_A);
//
//    kernel_A(1, stream_A);
//
//    cleanup_A();
//
//  } else if (strncmp("2", argv[1], 1) == 0) {
//    hipStream_t stream_B;
//    hipStreamCreate(&stream_B);
//
//    // grab kernel launch and exit function calls from benchmark A and B
//    std::function<int(const int, hipStream_t &)> kernel_B;
//    std::function<void(void)> cleanup_B;
//
//    invoke(B_str, argc_B, &(argv[B_idx]), kernel_B, cleanup_B);
//
//    kernel_B(1, stream_B);
//
//    cleanup_B();
//
//  } else {
//    // run both
//    // create two different cuda streams
//    hipStream_t stream_A, stream_B;
//
//    hipStreamCreate(&stream_A);
//    hipStreamCreate(&stream_B);
//
//    // grab kernel launch and exit function calls from benchmark A and B
//    std::function<int(const int, hipStream_t &)> kernel_A, kernel_B;
//    std::function<void(void)> cleanup_A, cleanup_B;
//
//    invoke(A_str, argc_A, &(argv[A_idx]), kernel_A, cleanup_A);
//    invoke(B_str, argc_B, &(argv[B_idx]), kernel_B, cleanup_B);
//
//    int iters = 5;
//
//    if (A_str.compare("+spmv") == 0 || B_str.compare("+spmv") == 0) {
//      iters = 30;
//      std::cout << "Launching 30 iters." << std::endl;
//    }
//
//    for (int i = 0; i < iters; i++) {
//      std::cout << "Launching one iteration" << std::endl;
//      kernel_A(1, stream_A);
//      kernel_B(1, stream_B);
//    }
//
//    cleanup_A();
//    cleanup_B();
//

//  }

}
